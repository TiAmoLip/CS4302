#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define OFFSET(row, col, ld) ((row) * (ld) + (col)) //ld是矩阵的列数
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

// A: 1*1280, B: 1280*5893, C: 1*5893
__global__ void Sgemm(float * A,float * B, float * C, int M, int N, int K) {
    
    const int BLOCK_SIZE_M = 1;// 一个block中处理的A矩阵行数,或者说一个block处理的A矩阵大小为BLOCK_SIZE_M*BLOCK_SIZE_K
    const int BLOCK_SIZE_N = 128;// 一个block中处理的B矩阵列数,或者说一个block处理的B矩阵大小为BLOCK_SIZE_K*BLOCK_SIZE_N
    const int BLOCK_SIZE_K = 8;// 一个block中处理的A矩阵列数和B矩阵行数,或者说一个block处理的C矩阵大小为BLOCK_SIZE_M*BLOCK_SIZE_N
    // 在上面的设置下，需要开启的线程数就是2048*2048/128/128=16*16=256
    // 在矩阵分块的基础上，我们对每一个block中的A和B矩阵继续进行分块，让每个线程处理rm*rn的子矩阵，那么一个block中的线程数就是BLOCK_SIZE_M*BLOCK_SIZE_N/THREAD_SIZE_Y/THREAD_SIZE_X
    const int TM = 1;
    const int TN = 4;
    
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int tid = ty * blockDim.x + tx;
    if (tid==0) printf("A[0][0] = %f, B[0][0] = %f\n", A[0], B[0]);
    __shared__ float s_a[BLOCK_SIZE_M][BLOCK_SIZE_K];
    __shared__ float s_b[BLOCK_SIZE_K][BLOCK_SIZE_N];
    float operator_b[TN];// 在最后计算的时候，将东西load到register，然后和s_a做乘法，实测从162变成158
    float r_c[TN] = {0.0f};
    int load_B_rows = tid >> 5;// 在block中当前线程在B应该load的行
    int load_B_cols = (tid & 31) << 2;// 在block中当前线程在B应该load的列
    // 算了，让所有线程都load A的值吧
    #pragma unroll
    for (int bk = 0; bk < (K+BLOCK_SIZE_K-1)/BLOCK_SIZE_K;++bk) {
        int load_A_cols = bk * BLOCK_SIZE_K;
        // 注意我们只开了32个线程，所以没办法一次就把s_b里的所有东西都load进来
        for (int i=0;i<BLOCK_SIZE_K;++i) {
            if (load_A_cols + i < K) {
                s_a[0][i] = A[OFFSET(0, load_A_cols + i, K)];
            } else {
                s_a[0][i] = 0.0f;
            }
        }
        // __syncthreads();
        // 接下来是一个循环将所有的B load进来，但是要注意边界检查
        // 但是这种做贡献的矩阵乘法写法，好像只要在最后数据写回的时候做边界检查就可以了。
        // 这里爆了一个cudaError: Misaligned address错误，原因是我在这里写的时候写到了B的边界外面去了
        // 就是这个循环的问题
        // cnm, 怎么有的float4正常用，有的就报错，服了，直接float吧
        int global_load_b_row_start = load_B_rows + bk * BLOCK_SIZE_K;        
        int global_load_b_col = bx*BLOCK_SIZE_N + load_B_cols;
        // 现在是32个线程，每个线程load1个float,此时load_B_rows = tid/128, load_B_cols = (tid%32)
        // 现在一个row要load4遍
        int current_load_B_col = tid << 2;
        #pragma unroll
        for (int i=0;i<BLOCK_SIZE_K;++i) {
            
            #pragma unroll
            for (int j=0;j<4;++j) {
                if (global_load_b_row_start + i < K && global_load_b_col + j < N) {
                    s_b[i][current_load_B_col+j] = B[OFFSET(global_load_b_row_start + i, global_load_b_col + j, N)];
                } else {
                    s_b[i][current_load_B_col+j] = 0.0f;
                }
            }
        }
        __syncthreads();
        // 我试一下load到寄存器里再做乘法会不会更快

        // 但是这里用float4更快
        #pragma unroll
        for (int k=0;k<BLOCK_SIZE_K;++k) {
            FETCH_FLOAT4(operator_b) = FETCH_FLOAT4(s_b[k][tx*TN]);
            #pragma unroll
            for (int n=0;n<TN;++n) {
                r_c[n] += s_a[0][k] * operator_b[n];
            }
        }
        // __syncthreads();
    }
    // 这个好奇怪啊，用float4反而更慢了
    #pragma unroll
    for (int j=0;j<TN;++j) {
        int global_c_col = bx*BLOCK_SIZE_N + tx*TN+j;
        if (global_c_col < N) {
            C[OFFSET(0, global_c_col, N)] = r_c[j];
        }
    }
}

// A: 5893 * 1280, B: (1280, 1), C: (5893,1)

__global__ void Sgemm_naive_stable1(float * A,float * B, float * C, int M, int N, int K) {
    // 76ms
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    float operator_a[4];
    float operator_b[4];
    if (row < M) {
        float sum = 0.0f;

        for (int i = 0; i < K/4; i++) {
            FETCH_FLOAT4(operator_a) = FETCH_FLOAT4(A[row*K+4*i]);
            FETCH_FLOAT4(operator_b) = FETCH_FLOAT4(B[4*i]);
            sum += operator_a[0] * operator_b[0] + operator_a[1] * operator_b[1] + operator_a[2] * operator_b[2] + operator_a[3] * operator_b[3];
        }
        C[row] = sum;
    }
}

__global__ void Sgemm_naive(float * A,float * B, float * C, int M, int N, int K) {
    // 93ms
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    float operator_a[2][4];
    float operator_b[2][4];
    if (row < M) {
        float sum = 0.0f;
        int write_idx = 0;
        FETCH_FLOAT4(operator_a[1-write_idx]) = FETCH_FLOAT4(A[row*K]);
        FETCH_FLOAT4(operator_b[1-write_idx]) = FETCH_FLOAT4(B[0]);
        for (int i = 0; i < K/4-1; i++) {
            int load_idx = 1-write_idx;
            FETCH_FLOAT4(operator_a[write_idx]) = FETCH_FLOAT4(A[row*K+(i+1)*4]);
            FETCH_FLOAT4(operator_b[write_idx]) = FETCH_FLOAT4(B[(i+1)*4]);
            sum += operator_a[load_idx][0] * operator_b[load_idx][0] + operator_a[load_idx][1] * operator_b[load_idx][1] + operator_a[load_idx][2] * operator_b[load_idx][2] + operator_a[load_idx][3] * operator_b[load_idx][3];
            write_idx = 1 - write_idx;
        }
        int load_idx = 1-write_idx;
        sum += operator_a[load_idx][0] * operator_b[load_idx][0] + operator_a[load_idx][1] * operator_b[load_idx][1] + operator_a[load_idx][2] * operator_b[load_idx][2] + operator_a[load_idx][3] * operator_b[load_idx][3];

        C[row] = sum;
    }
}

void read_numpy_data(float *A, float *B, float *C, int M, int N, int K) {
    FILE *fp = fopen("new_A.bin", "rb");
    fread(A, sizeof(float), M * K, fp);
    fclose(fp);
    fp = fopen("new_B.bin", "rb");
    fread(B, sizeof(float), K * N, fp);
    fclose(fp);
    fp = fopen("new_C.bin", "rb");
    fread(C, sizeof(float), M * N, fp);
    fclose(fp);
}

void check_result(float *C_cpu, float *C_cuda, int M, int N) {
    for (int i=0;i<M;++i) {
        for (int j=0;j<N;++j) {
            if (abs(C_cpu[i*N+j] - C_cuda[i*N+j]) > 1e-2) {
                printf("Error: C_cpu[%d][%d] = %f, C_cuda[%d][%d] = %f\n", i, j, C_cpu[i*N+j], i, j, C_cuda[i*N+j]);
                return;
            }
        }
    }
    printf("Check result: Correct\n");
}

float testPerformance(
    void (*gpuSgemm) (float *, float *, float *, const int, const int, const int),
    dim3 gridDim, dim3 blockDim, const int M, const int N, const int K, const int repeat) {

    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    for (int i = 0; i < repeat; i++)
        gpuSgemm<<<gridDim, blockDim>>>(d_a, d_b, d_c, M, N, K);
    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000.0 / repeat;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return sec;
}

float testCublasPerformance(const int M, const int N, const int K, const int repeat) {

    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    float cublas_alpha = 1.0;
    float cublas_beta = 0;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    for (int i = 0; i < repeat; i++) {
        //cublasSgemm(cublas_handle, CUBLAS_OP_T, CUBLAS_OP_T, M, N, K, &cublas_alpha, d_a, K, d_b, N, &cublas_beta, d_c, M);
        hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &cublas_alpha, d_b, N, d_a, K, &cublas_beta, d_c, N);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000.0 / repeat;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return sec;
}

int main() {
    const int M = 5893, N = 1, K = 1280;
    // const int M = 1, N = 5893, K = 1280;
    float *A_cpu = (float *)malloc(M * K * sizeof(float));
    float *B_cpu = (float *)malloc(K * N * sizeof(float));
    float *C_cpu = (float *)malloc(M * N * sizeof(float));
    float *result_cuda = (float *)malloc(M * N * sizeof(float));
    int threadsPerBlock = 128;
    dim3 DimGrid((M+threadsPerBlock-1)/threadsPerBlock, 1, 1);
    dim3 DimBlock(threadsPerBlock, 1, 1);
    read_numpy_data(A_cpu, B_cpu, C_cpu, M, N, K);
    float *cuda_A, *cuda_B, *cuda_C;
    hipMalloc(&cuda_A, M * K * sizeof(float));
    hipMalloc(&cuda_B, K * N * sizeof(float));
    hipMalloc(&cuda_C, M * N * sizeof(float));
    hipMemcpy(cuda_A, A_cpu, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cuda_B, B_cpu, K * N * sizeof(float), hipMemcpyHostToDevice);
    Sgemm_naive<<<DimGrid, DimBlock>>>(cuda_A, cuda_B, cuda_C, M, N, K);
    hipMemcpy(result_cuda, cuda_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
    check_result(C_cpu, result_cuda, M, N);
    float sec = testPerformance(Sgemm_naive, DimGrid, DimBlock, M, N, K, 100);
    printf("Kernel time cost: %.6f ms\n", sec);
    sec = testCublasPerformance(M, N, K, 100);
    printf("Cublas time cost: %.6f ms\n", sec);
    hipError_t err = hipGetLastError();  
    if (err != hipSuccess) {  
        printf("CUDA error: %s\n", hipGetErrorString(err));  
    }
}